#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// EDGE DETECTION -- Jona Cappelle -- Jonas Bolle
////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// own includes
#include "iostream"
#include "cstdlib"
#include "time.h"	// timing on cpu
#include "lodepng.h" // PNG afbeelding inlezen

extern "C"

void decodeOneStep(const char* filename) {
	unsigned error;
	unsigned char* image = 0;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	/*use image here*/

	free(image);
}

void encodeOneStep(const char* filename, const unsigned char* image, unsigned width, unsigned height) {
	/*Encode the image*/
	unsigned error = lodepng_encode32_file(filename, image, width, height);

	/*if there's an error, display it*/
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

////////////////////////////////////////////////////////////////////////////////
// KERNEL GRAYSCALE
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;

// GPU
__global__ void edge(unsigned char* orig, unsigned char* result,unsigned width,unsigned height)
{

	int x = (threadIdx.x + blockIdx.x * blockDim.x)*4;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    float dx, dy;
    width=4*width;
    if( x > 0 && y > 0 && x < (width-1) && y < (height-1)) {
        dx = (-1* orig[(y-1)*width + (x-4)]) + (-2*orig[y*width+(x-4)]) + (-1*orig[(y+1)*width+(x-4)]) +
             (    orig[(y-1)*width + (x+4)]) + ( 2*orig[y*width+(x+4)]) + (   orig[(y+1)*width+(x+4)]);
        dy = (    orig[(y-1)*width + (x-4)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+4)]) +
             (-1* orig[(y+1)*width + (x-4)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+4)]);
        result[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 1] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 2] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 3] = 255;
    }

}

// CPU
void edge_cpu(unsigned char* image, unsigned width, unsigned height)
{
	printf("test1");

	for(int j=0; j < (width*height*4); j+=4)
	{
		image[j] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+1] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+2] = (image[j]+image[j+1]+image[j+2])/3;

		if((image[j+4] - image[j]) > 5 )
		{
			image[j] = image[j+1] =image[j+2]= 255;
		}else{
			image[j] = image[j+1] =image[j+2]= 0;
		}

	}
	printf("test2");
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
	////////////////////////////////////////////////////////////////////////////////
	// Load PNG file
	////////////////////////////////////////////////////////////////////////////////
	float millis = 0;
	unsigned char *image_in, *image_out, *image_in_dev, *image_out_dev;
	unsigned width, height, width_dev, height_dev;

	const char* filename = "test.png";

	unsigned error;
	unsigned char* image = 0;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	// allocate arrays on host
	image_in = (unsigned char *)malloc(width*height*4 * sizeof(char));
	image_out = (unsigned char *)malloc(width*height*4 * sizeof(char));


	FILE *f = fopen("data.csv", "w");

//	StopWatchInterface *timer = 0;
//	sdkCreateTimer(&timer);
//	sdkResetTimer(&timer);
//	sdkStartTimer(&timer);

//	edge_cpu(image, width, height);

//	sdkStopTimer(&timer);
//	printf("Tijd: %f\n", sdkGetTimerValue(&timer));
//	sdkDeleteTimer(&timer);

	// Grayscale on CPU
	for(int j=0; j < (width*height*4); j+=4)
	{
		image[j] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+1] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+2] = (image[j]+image[j+1]+image[j+2])/3;
	}

	// Allocate arrays on device
	hipMalloc((void **)&image_in_dev, width*height*4 * sizeof(char));
	hipMalloc((void **)&image_out_dev, width*height*4 * sizeof(char));

	hipMemcpy(image_in_dev, image, width*height*4*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(image_out_dev, image_out, width*height*4*sizeof(char), hipMemcpyHostToDevice);

	// Record time on GPU with cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	///////////////////////////
	// Choose Blocksize & nBlock in 2D
	dim3 BLOCKSIZE(64,16);
	dim3 nBlocks(ceil(width/64),ceil(height/16));
	///////////////////////////

	hipEventRecord(start);
	edge <<< nBlocks, BLOCKSIZE >>> ( image_in_dev, image_out_dev, width, height );
	hipEventRecord(stop);

	hipMemcpy(image_in, image_in_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(image_out, image_out_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);

	printf("Tijd op GPU: %f\n", millis);

//	fprintf(f, "%d,%f\n", BLOCKSIZE, millis);

	fclose(f);

	const char* output_filename = "output.png";
	encodeOneStep(output_filename, image_out, width, height);

	free(image_in);
	free(image_out);

	hipFree(image_in_dev);
	hipFree(image_out_dev);

	printf("Done!");

	return 0;
}
