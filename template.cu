#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// ADD / INVERT -- Jona Cappelle -- Jonas Bolle
////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// eigen includes
#include "iostream"
#include "cstdlib"
#include "time.h"	// timing on cpu

extern "C"
#define ARRAYSIZE 100000000 // Is also the number of threads that will be used


// HELPER FUNCTIONS
void init_array(int *a)
{
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		a[i] = i;
	}
}


////////////////////////////////////////////////////////////////////////////////
// KERNEL ADD
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;

// GPU
__global__ void add(int *a, int *b, int *out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < ARRAYSIZE)
	{
		out[idx] = a[idx] + b[idx];
	}
}

// CPU
void cpu_add(int *a, int *b, int *out)
{
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		out[i] = a[i] + b[i];
	}
}

////////////////////////////////////////////////////////////////////////////////
// KERNEL INVERT
////////////////////////////////////////////////////////////////////////////////

// GPU
__global__ void invert(int *a, int *out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < ARRAYSIZE)
	{
		out[idx] = a[ARRAYSIZE - 1 - idx];
	}
}

// CPU
void cpu_invert(int *a, int *out)
{
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		out[i] = a[ARRAYSIZE - 1 - i];
	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{

	//declare variables
	int *a_host, *b_host, *out_host;
	int *a_dev, *b_dev, *out_dev;


	//allocate arrays on host
	a_host = (int *)malloc(ARRAYSIZE * sizeof(int));

	b_host = (int *)malloc(ARRAYSIZE * sizeof(int));
	out_host = (int *)malloc(ARRAYSIZE * sizeof(int));

	init_array(a_host);
	init_array(b_host);

	//allocate arrays on device
	hipMalloc((void **)&a_dev, ARRAYSIZE * sizeof(int));
	hipMalloc((void **)&b_dev, ARRAYSIZE * sizeof(int));
	hipMalloc((void **)&out_dev, ARRAYSIZE * sizeof(int));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Timer on CPU
//	clock_t start, end;
//	double cpu_time_used;

//	 Initialize data file where the timing results will be stored
	 FILE *f = fopen("data.csv", "w");

	for (int BLOCKSIZE = 1; BLOCKSIZE < 300; BLOCKSIZE++)
	{
		float millis = 0;
		// Calculate amount of blocks needed
		int nBlocks = ARRAYSIZE / BLOCKSIZE + (ARRAYSIZE % BLOCKSIZE == 0 ? 0 : 1);
		printf("Nblocks: %i", nBlocks);

		// Start timer
		StopWatchInterface *timer = 0;
		sdkCreateTimer(&timer);
		sdkStartTimer(&timer);
//		hipEventRecord(start);
		//Step 1: Copy data to GPU memory
		hipMemcpy(a_dev, a_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(b_dev, b_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(out_dev, out_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);

		////////////////////////////////////////////////////////////////////////////////
		// GPU -- comment / uncomment to run 'ADD' / 'INVERT'
		////////////////////////////////////////////////////////////////////////////////

//		add<<<nBlocks, BLOCKSIZE>>>(a_dev, b_dev, out_dev);
		invert <<< nBlocks, BLOCKSIZE >>> ( a_dev, out_dev );
//		hipEventRecord(stop);

		////////////////////////////////////////////////////////////////////////////////
		// CPU -- comment / uncomment to run 'ADD' / 'INVERT'
		////////////////////////////////////////////////////////////////////////////////

//		start = clock();
//		cpu_add( a_host, b_host, out_host);
//		cpu_invert ( a_host, out_host );
//		end = clock();
//		cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
//		printf("%f", cpu_time_used);

		//Step 4: Retrieve result
		hipMemcpy(a_host, a_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(b_host, b_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(out_host, out_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);

//		hipEventSynchronize(stop);
//		hipEventElapsedTime(&millis, start, stop);

		// Stop timer
		sdkStopTimer(&timer);

		// Print time to console
		printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
//		printf("Processing time: %f (ms)\n", millis);

		// Write timing results to file
		fprintf(f, "%d,%f\n", BLOCKSIZE, sdkGetTimerValue(&timer));
//		fprintf(f, "%f\n", sdkGetTimerValue(&timer));
//		fprintf(f, "%d,%f\n", BLOCKSIZE, millis);

		// Verwijder timer
		sdkDeleteTimer(&timer);

	} //End for

	// Close the file
	fclose(f);


	// Free up the used memory
	free(a_host);
	free(b_host);
	free(out_host);
	hipFree(a_dev);
	hipFree(b_dev);
	hipFree(out_dev);

	return 0;
}
