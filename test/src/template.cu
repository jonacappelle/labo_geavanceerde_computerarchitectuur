#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// ADD / INVERT -- Jona Cappelle -- Jonas Bolle
////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// eigen includes
#include "iostream"
#include "cstdlib"

extern "C"
#define ARRAYSIZE 100 // Is also the number of threads that will be used
#define BLOCKSIZE

////////////////////////////////////////////////////////////////////////////////
// KERNEL ADD
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;
int nBlocks;
// GPU
__global__ void add(int *a, int *b, int *out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < ARRAYSIZE)
	{
		out[idx] = a[idx] + b[idx];
	}
}

// CPU
void add(int *a, int *b, int *out)
{
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		out[i] = a[i] + b[i];
	}
}

////////////////////////////////////////////////////////////////////////////////
// KERNEL INVERT
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;
int nBlocks;

// GPU
__global__ void invert(int *a, int *out)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < ARRAYSIZE)
	{
		out[idx] = a[ARRAYSIZE - idx]
	}
}

// CPU
void invert(int *a, int *out)
{
	for (int i = 0; i < ARRAYSIZE; i++)
	{
		out[i] = a[ARRAYSIZE - i];
	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{

	//declare variables
	int *a_host, *b_host, *out_host;
	int *a_dev, *b_dev, *out_dev;

	//allocate arrays on host
	a_host = (int *)malloc(ARRAYSIZE * sizeof(int));

	b_host = (int *)malloc(ARRAYSIZE * sizeof(int));
	out_host = (int *)malloc(ARRAYSIZE * sizeof(int));

	//allocate arrays on device
	hipMalloc((void **)&a_dev, ARRAYSIZE * sizeof(int));
	hipMalloc((void **)&b_dev, ARRAYSIZE * sizeof(int));
	hipMalloc((void **)&out_dev, ARRAYSIZE * sizeof(int));

	//Step 1: Copy data to GPU memory
	hipMemcpy(a_dev, a_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(out_dev, out_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);

	// Initialize data file where the timing results will be stored
	FILE *f = fopen("data.csv", "w");

	for (BLOCKSIZE = 1; BLOCKSIZE < 1024; BLOCKSIZE++)
	{
		// Calculate amount of blocks needed
		int nBlocks = ARRAYSIZE / BLOCKSIZE + (ARRAYSIZE % BLOCKSIZE == 0 ? 0 : 1);
		printf("Nblocks: %i", nBlocks);

		// Start timer
		StopWatchInterface *timer = 0;
		sdkCreateTimer(&timer);
		sdkStartTimer(&timer);

		////////////////////////////////////////////////////////////////////////////////
		// GPU -- comment / uncomment to run 'ADD' / 'INVERT'
		////////////////////////////////////////////////////////////////////////////////
		add<<<nBlocks, BLOCKSIZE>>>(a_dev, b_dev, out_dev);
		// invert <<< nBlocks, BLOCKSIZE >>> ( a_dev, out_dev );

		////////////////////////////////////////////////////////////////////////////////
		// CPU -- comment / uncomment to run 'ADD' / 'INVERT'
		////////////////////////////////////////////////////////////////////////////////
		// add( a_dev, b_dev, out_dev );
		// invert ( a_dev, out_dev );

		// Stop timer
		sdkStopTimer(&timer);
		printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));

		// Write timing results to file
		fprintf(f, "%d,%f\n", BLOCKSIZE, sdkGetTimerValue(&timer));

		// Verwijder timer
		sdkDeleteTimer(&timer);

	} //End for

	// Close the file
	fclose(f);

	//Step 4: Retrieve result
	hipMemcpy(a_host, a_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b_host, b_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(out_host, out_dev, ARRAYSIZE * sizeof(int), hipMemcpyDeviceToHost);

	// Free up the used memory
	free(a_host);
	free(b_host);
	free(out_host);
	hipFree(a_dev);
	hipFree(b_dev);
	hipFree(out_dev);

	return 0;
}
