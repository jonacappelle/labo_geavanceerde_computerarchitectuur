#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// GRAYSCALE - EDGE DETECTION -- Jona Cappelle -- Jonas Bolle
////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// own includes
#include "iostream"
#include "cstdlib"
#include "time.h"	// timing on cpu
#include "lodepng.h" // PNG afbeelding inlezen

extern "C"


////////////////////////////////////////////////////////////////////////////////
// SELECT GPU - CPU TIMING
// #define GPU
//#define CPU
////////////////////////////////////////////////////////////////////////////////
// RUN ADD - INV
// #define ADD
// #define INV
////////////////////////////////////////////////////////////////////////////////

// Helper function



void decodeOneStep(const char* filename) {
	unsigned error;
	unsigned char* image = 0;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	/*use image here*/

	free(image);
}

void encodeOneStep(const char* filename, const unsigned char* image, unsigned width, unsigned height) {
	/*Encode the image*/
	unsigned error = lodepng_encode32_file(filename, image, width, height);

	/*if there's an error, display it*/
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

////////////////////////////////////////////////////////////////////////////////
// KERNEL GRAYSCALE
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;

// GPU
//__global__ void grayscale(unsigned char* image, unsigned char* grayImage,unsigned width,unsigned height)
//{
//	int absolute_position_x =(blockIdx.x * blockDim.x) + threadIdx.x;
//	int absolute_position_y = (blockIdx.y * blockDim.y) + threadIdx.y;
//	if(absolute_position_x >= width || absolute_position_y >= height){
//		return;
//	}
//
//	float channelSum = .299f * image[absolute_position_x + absolute_position_y * width]
//									 + .587f * image[(absolute_position_x + absolute_position_y * width)+1]
//									 + .114f * image[(absolute_position_x + absolute_position_y * width)+2];
//	grayImage[absolute_position_x + absolute_position_y * width] = channelSum;
//}

__global__ void grayscale(unsigned char* image, unsigned char* grayImage,unsigned width,unsigned height)
{
	int j = (blockIdx.x*blockDim.x + threadIdx.x)*4;

	if(j < width*height*4)
	{
		grayImage[j] = (image[j]+image[j+1]+image[j+2])/3;
		grayImage[j+1] = (image[j]+image[j+1]+image[j+2])/3;
		grayImage[j+2] = (image[j]+image[j+1]+image[j+2])/3;
		grayImage[j+3] = 255;
	}

}

// CPU
void grayscale_cpu(unsigned char* image, unsigned width, unsigned height)
{
	printf("test1");

	for(int j=0; j < (width*height*4); j+=4)
	{
		image[j] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+1] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+2] = (image[j]+image[j+1]+image[j+2])/3;
	}
	printf("test2");
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{

	printf("testest");


	////////////////////////////////////////////////////////////////////////////////
	// Load PNG file
	////////////////////////////////////////////////////////////////////////////////
	float millis = 0;
	unsigned char *image_in, *image_out, *image_in_dev, *image_out_dev;
	unsigned width, height, width_dev, height_dev;

	printf("test-2");

	const char* filename = "test.png";

	unsigned error;
	unsigned char* image = 0;


	printf("test-1");



	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));



	// allocate arrays on host
	image_in = (unsigned char *)malloc(width*height*4 * sizeof(char));
	image_out = (unsigned char *)malloc(width*height*4 * sizeof(char));


FILE *f = fopen("data.csv", "w");

for (int BLOCKSIZE = 1; BLOCKSIZE < 300; BLOCKSIZE++)
{
	int nBlocks = (width*height*4) / BLOCKSIZE + ((width*height*4) % BLOCKSIZE == 0 ? 0 : 1);
	printf("nBlocks: %d", nBlocks);
	// image wordt goed geprint
//	for(int i=0; i<(width*height*4); i+=4)
//	{
//	printf("%u %u %u %u\n", image[i], image[i+1], image[i+2], image[i+3]);
//	}


//	StopWatchInterface *timer = 0;
//	sdkCreateTimer(&timer);
//	sdkResetTimer(&timer);
//	sdkStartTimer(&timer);

//	grayscale_cpu(image, width, height);

//	sdkStopTimer(&timer);
//	printf("Tijd: %f\n", sdkGetTimerValue(&timer));
//	sdkDeleteTimer(&timer);


	// 	// allocate arrays on device
	hipMalloc((void **)&image_in_dev, width*height*4 * sizeof(char));
	hipMalloc((void **)&image_out_dev, width*height*4 * sizeof(char));



	hipMemcpy(image_in_dev, image, width*height*4*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(image_out_dev, image_out, width*height*4*sizeof(char), hipMemcpyHostToDevice);

//	unsigned *width_1 = &width;
//	hipMemcpy(width_dev, width_1, sizeof(unsigned), hipMemcpyHostToDevice);
//	hipMemcpy(height_dev, &height, sizeof(unsigned), hipMemcpyHostToDevice);

	printf("Dit is van de CPU");
	for(int i=100*4; i<4*120; i+=4)
	{
	printf("%u %u %u %u\n", image[i], image[i+1], image[i+2], image[i+3]);
	}


	// Record time on GPU with cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);




	hipEventRecord(start);
	grayscale <<< nBlocks, BLOCKSIZE >>> ( image_in_dev, image_out_dev, width, height );
	hipEventRecord(stop);

	hipMemcpy(image_in, image_in_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(image_out, image_out_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);
//	hipMemcpy(width, width_dev, sizeof(unsigned), hipMemcpyDeviceToHost);
//	hipMemcpy(height, height_dev, sizeof(unsigned), hipMemcpyDeviceToHost);

	printf("Dit is van de GPU");
	for(int i=100*4; i<4*120; i+=4)
		{
		printf("%u %u %u %u\n", image_out[i], image_out[i+1], image_out[i+2], image_out[i+3]);
		}

	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);

	printf("Tijd op GPU: %f\n", millis);

	fprintf(f, "%d,%f\n", BLOCKSIZE, millis);

//	hipEventDestroy(stop);
//	hipEventDestroy(start);

}
fclose(f);

	const char* output_filename = "output.png";
	encodeOneStep(output_filename, image_out, width, height);


	free(image_in);
	free(image_out);
//	free(width);
//	free(height);
	hipFree(image_in_dev);
	hipFree(image_out_dev);
//	hipFree(width_dev);
//	hipFree(height_dev);

	printf("Done!");

	return 0;
}
