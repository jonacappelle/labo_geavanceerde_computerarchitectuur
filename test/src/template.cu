#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

//eigen includes
#include "iostream"
#include "cstdlib"


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);


// KERNEL





#define ARRAYSIZE 100
//#define BLOCKSIZE
int BLOCKSIZE;

int nBlocks;


// Kernel
__global__ void add(int *a, int *b, int *out){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx < ARRAYSIZE){
		out[idx] = a[idx] + b[idx];
	}
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{

	//declare variables
		int *a_host, *b_host, *out_host;
		int *a_dev, *b_dev, *out_dev;


		//allocate arrays on host
		a_host = (int*)malloc(ARRAYSIZE * sizeof(int));

		b_host = (int *)malloc(ARRAYSIZE * sizeof(int));
		out_host = (int *)malloc(ARRAYSIZE * sizeof(int));

		//allocate arrays on device
		hipMalloc((void **) &a_dev, ARRAYSIZE * sizeof(int));
		hipMalloc((void **) &b_dev, ARRAYSIZE * sizeof(int));
		hipMalloc((void **) &out_dev, ARRAYSIZE * sizeof(int));


	//rest of program (Other 4 steps go here)

	//Step 1: Copy data to GPU memory

	hipMemcpy(a_dev, a_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(out_dev, out_host, ARRAYSIZE * sizeof(int), hipMemcpyHostToDevice);


	FILE *f = fopen("data.csv", "w");

for ( BLOCKSIZE = 1; BLOCKSIZE < 1024; BLOCKSIZE++ )
{
	int nBlocks = ARRAYSIZE/BLOCKSIZE + (ARRAYSIZE%BLOCKSIZE == 0?0:1);
	printf("Nblocks: %i", nBlocks);

	// Measure time
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

	add <<< nBlocks, BLOCKSIZE >>> (a_dev, b_dev, out_dev);

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));


    fprintf(f, "%d,%f\n", BLOCKSIZE, sdkGetTimerValue(&timer));



    sdkDeleteTimer(&timer);

}//End for

	fclose(f);



	//Step 4: Retrieve result
	hipMemcpy(a_host, a_dev, ARRAYSIZE * sizeof(int), 	hipMemcpyDeviceToHost);
	hipMemcpy(b_host, b_dev, ARRAYSIZE * sizeof(int), 	hipMemcpyDeviceToHost);
	hipMemcpy(out_host, out_dev, ARRAYSIZE * sizeof(int), 	hipMemcpyDeviceToHost);


	//end of  program
	//cleanup: VERY IMPORTANT!!!
	free(a_host); free(b_host); free(out_host); hipFree(a_dev); hipFree(b_dev); hipFree(out_dev);

return 0;
}
