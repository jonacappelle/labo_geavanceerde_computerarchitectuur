#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// EDGE DETECTION -- Jona Cappelle -- Jonas Bolle
////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

// own includes
#include "iostream"
#include "cstdlib"
#include "time.h"	// timing on cpu
#include "lodepng.h" // PNG afbeelding inlezen

extern "C"

void decodeOneStep(const char* filename) {
	unsigned error;
	unsigned char* image = 0;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	/*use image here*/

	free(image);
}

void encodeOneStep(const char* filename, const unsigned char* image, unsigned width, unsigned height) {
	/*Encode the image*/
	unsigned error = lodepng_encode32_file(filename, image, width, height);

	/*if there's an error, display it*/
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

////////////////////////////////////////////////////////////////////////////////
// KERNEL EDGE DETECTION
////////////////////////////////////////////////////////////////////////////////
int BLOCKSIZE;

// GPU
__global__ void edge(unsigned char* orig, unsigned char* result,unsigned width,unsigned height)
{

	int x = (threadIdx.x + blockIdx.x * blockDim.x)*4;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    float dx, dy;
    width=4*width;
    if( x > 0 && y > 0 && x < (width-1) && y < (height-1)) {
        dx = (-1* orig[(y-1)*width + (x-4)]) + (-2*orig[y*width+(x-4)]) + (-1*orig[(y+1)*width+(x-4)]) +
             (    orig[(y-1)*width + (x+4)]) + ( 2*orig[y*width+(x+4)]) + (   orig[(y+1)*width+(x+4)]);
        dy = (    orig[(y-1)*width + (x-4)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+4)]) +
             (-1* orig[(y+1)*width + (x-4)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+4)]);
        result[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 1] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 2] = sqrt( (dx*dx) + (dy*dy) );
        result[y*width + x + 3] = 255;
    }

}

// CPU
void edge_cpu(unsigned char* image,unsigned char* image_out, unsigned width, unsigned height)
{

	for(int j=0; j < (width*height*4); j+=4)
    {
        image[j] = (image[j]+image[j+1]+image[j+2])/3;
        image[j+1] = (image[j]+image[j+1]+image[j+2])/3;
        image[j+2] = (image[j]+image[j+1]+image[j+2])/3;
    }
    int dx,dy,val;
	 int sobel_x[3][3] =
        { { -1, 0, 1 },
          { -2, 0, 2 },
          { -1, 0, 1 } };

    int sobel_y[3][3] =
        { { -1, -2, -1 },
          { 0,  0,  0 },
          { 1,  2,  1 } };

    for (int x=1; x < (width-1); x++)
      {
         for (int y=1; y < (height-1); y++)
            {

                dx = (sobel_x[0][0] * image[width*4 * (y-1) + (x-1)*4])
                        + (sobel_x[0][1] * image[width*4 * (y-1) +  x*4   ])
                        + (sobel_x[0][2] * image[width*4 * (y-1) + (x+1)*4])
                        + (sobel_x[1][0] * image[width*4 *  y    + (x-1)*4])
                        + (sobel_x[1][1] * image[width*4 *  y    +  x *4  ])
                        + (sobel_x[1][2] * image[width*4 *  y    + (x+1)*4])
                        + (sobel_x[2][0] * image[width*4 * (y+1) + (x-1)*4])
                        + (sobel_x[2][1] * image[width*4 * (y+1) +  x *4  ])
                        + (sobel_x[2][2] * image[width*4 * (y+1) + (x+1)*4]);

                dy = (sobel_y[0][0] * image[width*4 * (y-1) + (x-1)*4])
                        + (sobel_y[0][1] * image[width*4 * (y-1) +  x *4  ])
                        + (sobel_y[0][2] * image[width*4 * (y-1) + (x+1)*4])
                        + (sobel_y[1][0] * image[width*4 *  y    + (x-1)*4])
                        + (sobel_y[1][1] * image[width*4 *  y    +  x *4  ])
                        + (sobel_y[1][2] * image[width*4 *  y    + (x+1)*4])
                        + (sobel_y[2][0] * image[width*4 * (y+1) + (x-1)*4])
                        + (sobel_y[2][1] * image[width*4 * (y+1) +  x *4  ])
                        + (sobel_y[2][2] * image[width*4 * (y+1) + (x+1)*4]);

                int val = (int)sqrt((dx * dx) + (dy * dy));

                if(val < 0) val = 0;
                if(val > 255) val = 255;

                image_out[width*4 * (y-1) + (x-1)*4] = val;
                image_out[width*4 * (y-1) + (x-1)*4+1] = val;
                image_out[width*4 * (y-1) + (x-1)*4+2] = val;
                image_out[width*4 * (y-1) + (x-1)*4+3] = 255;
            }
      }
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
	////////////////////////////////////////////////////////////////////////////////
	// Load PNG file
	////////////////////////////////////////////////////////////////////////////////
	float millis = 0;
	unsigned char *image_in, *image_out, *image_in_dev, *image_out_dev;
	unsigned width, height, width_dev, height_dev;

	const char* filename = "test.png";

	unsigned error;
	unsigned char* image = 0;

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

	// allocate arrays on host
	image_in = (unsigned char *)malloc(width*height*4 * sizeof(char));
	image_out = (unsigned char *)malloc(width*height*4 * sizeof(char));


	FILE *f = fopen("data.csv", "w");

//	StopWatchInterface *timer = 0;
//	sdkCreateTimer(&timer);
//	sdkResetTimer(&timer);
//	sdkStartTimer(&timer);

//	edge_cpu(image, image_out, width, height);

//	sdkStopTimer(&timer);
//	printf("Tijd: %f\n", sdkGetTimerValue(&timer));
//	sdkDeleteTimer(&timer);

	// Grayscale on CPU
	for(int j=0; j < (width*height*4); j+=4)
	{
		image[j] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+1] = (image[j]+image[j+1]+image[j+2])/3;
		image[j+2] = (image[j]+image[j+1]+image[j+2])/3;
	}

	// Allocate arrays on device
	hipMalloc((void **)&image_in_dev, width*height*4 * sizeof(char));
	hipMalloc((void **)&image_out_dev, width*height*4 * sizeof(char));

	hipMemcpy(image_in_dev, image, width*height*4*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(image_out_dev, image_out, width*height*4*sizeof(char), hipMemcpyHostToDevice);

	// Record time on GPU with cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	///////////////////////////
	// Choose Blocksize & nBlock in 2D
	dim3 BLOCKSIZE(64,16);
	dim3 nBlocks(ceil(width/64),ceil(height/16));
	///////////////////////////

	hipEventRecord(start);
	edge <<< nBlocks, BLOCKSIZE >>> ( image_in_dev, image_out_dev, width, height );
	hipEventRecord(stop);

	hipMemcpy(image_in, image_in_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(image_out, image_out_dev, width*height*4*sizeof(char), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);

	printf("Tijd op GPU: %f\n", millis);

//	fprintf(f, "%d,%f\n", BLOCKSIZE, millis);

	fclose(f);

	const char* output_filename = "output.png";
	encodeOneStep(output_filename, image_out, width, height);

	free(image_in);
	free(image_out);

	hipFree(image_in_dev);
	hipFree(image_out_dev);

	printf("Done!");

	return 0;
}
